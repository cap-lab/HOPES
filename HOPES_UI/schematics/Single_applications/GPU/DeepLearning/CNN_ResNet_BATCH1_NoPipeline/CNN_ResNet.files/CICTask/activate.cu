#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

//#include "hip/hip_runtime.h"

extern "C" {
#include "hip/hip_runtime.h"
#include "activate.h"
}


__device__ float _relu_gpu(float x) {return x*(x>0);}
__device__ float _relu_differential_gpu(float x) {return (x>0) ? x : .1*x;}

__device__ float _leaky_relu_gpu(float x) {return (x>0) ? x : .1*x;}
__device__ float _leaky_relu_differential_gpu(float x) {return (x>0) ? 1 : .1;}

__device__ float _logistic_gpu(float x){return 1./(1. + exp(-x));}
__device__ float _logistic_differential_gpu(float x){return (1-x)*x;}



__global__ void relu_kernel(float *x, int n) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] = _relu_gpu(x[i]);
    }
}


__global__ void leaky_kernel(float *x, int n) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] = _leaky_relu_gpu(x[i]);
    }
}

__global__ void logistic_kernel(float *x, int n) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] = _logistic_gpu(x[i]);
    }
}

__global__ void gradient_relu_kernel(float *x, int n, float *delta) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        delta[i] *= _relu_differential_gpu(x[i]);
    }
}

__global__ void gradient_leaky_kernel(float *x, int n, float *delta) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        delta[i] *= _leaky_relu_differential_gpu(x[i]);
    }
}

__global__ void gradient_logistic_kernel(float *x, int n, float *delta) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        delta[i] *= _logistic_differential_gpu(x[i]);
    }
}

extern "C" void relu_gpu(float *x, int n) {
    hipStream_t stream;
    hipdnnGetStream(cudnn_handle(), &stream);
    relu_kernel<<<cuda_gridsize(n), BLOCK, 0, stream>>>(x, n);
}

extern "C" void leaky_relu_gpu(float *x, int n) {
    hipStream_t stream;
    hipdnnGetStream(cudnn_handle(), &stream);
    leaky_kernel<<<cuda_gridsize(n), BLOCK, 0>>>(x, n);
    //leaky_kernel<<<cuda_gridsize(n), BLOCK, 0, stream>>>(x, n);
	check_error(hipSuccess);
}

extern "C" void logistic_gpu(float *x, int n) {
    logistic_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n);
}

extern "C" void relu_grad_gpu(float *x, int n, float *delta) {
    gradient_relu_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, delta);
}

extern "C" void leaky_relu_grad_gpu(float *x, int n, float *delta) {
    gradient_leaky_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, delta);
}

extern "C" void logistic_grad_gpu(float *x, int n, float *delta) {
    gradient_logistic_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, delta);
}
